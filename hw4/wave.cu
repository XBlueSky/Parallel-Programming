/**********************************************************************
 * DESCRIPTION:
 *   Serial Concurrent Wave Equation - C Version
 *   This program implements the concurrent wave equation
 *********************************************************************/

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>

#define MAXPOINTS 1000000
#define MAXSTEPS 1000000
#define MINPOINTS 20
#define PI 3.14159265

void check_param(void);

void update (void);
void printfinal (void);

int nsteps,                 	/* number of time steps */
    tpoints, 	     		/* total points along string */
    rcode;                  	/* generic return code */
float  values[MAXPOINTS+2], 	/* values at time t */
       oldval[MAXPOINTS+2], 	/* values at time (t-dt) */
       newval[MAXPOINTS+2]; 	/* values at time (t+dt) */


/**********************************************************************
 *	Checks input values from parameters
 *********************************************************************/
void check_param(void)
{
   char tchar[20];

   /* check number of points, number of iterations */
   while ((tpoints < MINPOINTS) || (tpoints > MAXPOINTS)) {
      printf("Enter number of points along vibrating string [%d-%d]: "
           ,MINPOINTS, MAXPOINTS);
      scanf("%s", tchar);
      tpoints = atoi(tchar);
      if ((tpoints < MINPOINTS) || (tpoints > MAXPOINTS))
         printf("Invalid. Please enter value between %d and %d\n", 
                 MINPOINTS, MAXPOINTS);
   }
   while ((nsteps < 1) || (nsteps > MAXSTEPS)) {
      printf("Enter number of time steps [1-%d]: ", MAXSTEPS);
      scanf("%s", tchar);
      nsteps = atoi(tchar);
      if ((nsteps < 1) || (nsteps > MAXSTEPS))
         printf("Invalid. Please enter value between 1 and %d\n", MAXSTEPS);
   }

   printf("Using points = %d, steps = %d\n", tpoints, nsteps);

}

// __device__ inline unsigned global_thread_id() {
//       return blockIdx.x * blockDim.x + threadIdx.x;
// }
/**********************************************************************
 *     Initialize points on line
 *********************************************************************/
 __global__ void init_line(float *gpuValue, float *gpuOldval, int tpoints)
{
    unsigned id = blockIdx.x * blockDim.x + threadIdx.x;
    /* Calculate initial values based on sine curve */
    gpuValue[id] = sin((2.0 * PI) * ((float)id / (float)(tpoints - 1))); 
    gpuOldval[id] = sin((2.0 * PI) * ((float)id / (float)(tpoints - 1))); 
}

/**********************************************************************
 *     Update all values along line a specified number of times
 *********************************************************************/
 __global__ void update(float *gpuValue, float *gpuOldval, float *gpuNewval, int nsteps, int tpoints)
{
   int i;
   unsigned id = blockIdx.x * blockDim.x + threadIdx.x;
   /* Update values for each time step */
   for (i = 1; i<= nsteps; i++) {
    /* Update points along line for this time step */
        /* global endpoints */
        if ((id == 0) || (id  == tpoints - 1))
            gpuNewval[id] = 0.0;
        else
            gpuNewval[id] = 1.82 * gpuValue[id] - gpuOldval[id];

        /* Update old values with new values */
        gpuOldval[id] = gpuValue[id];
        gpuValue[id] = gpuNewval[id];
   }
}

/**********************************************************************
 *     Print final results
 *********************************************************************/
void printfinal()
{
   int i;

   for (i = 0; i < tpoints; i++) {
      printf("%6.4f ", values[i]);
      if (i%10 == 0)
         printf("\n");
   }
}

/**********************************************************************
 *	Main program
 *********************************************************************/
int main(int argc, char *argv[])
{
	sscanf(argv[1],"%d",&tpoints);
	sscanf(argv[2],"%d",&nsteps);
    check_param();
    float *gpuValue, *gpuOldval, *gpuNewval;

    hipMalloc(&gpuValue, sizeof(values));
    hipMalloc(&gpuOldval, sizeof(values));
    hipMalloc(&gpuNewval, sizeof(values));

	printf("Initializing points on the line...\n");
    init_line<<<((tpoints + 1023) >> 10), 1024>>>(gpuValue, gpuOldval, tpoints);
    hipMemcpy(values, gpuValue, sizeof(values), hipMemcpyDeviceToHost);
    hipMemcpy(oldval, gpuOldval, sizeof(values), hipMemcpyDeviceToHost);

    printf("Updating all points for all time steps...\n");
	update<<<((tpoints + 1023) >> 10), 1024>>>(gpuValue, gpuOldval, gpuNewval, nsteps, tpoints);
    hipMemcpy(values, gpuValue, sizeof(values), hipMemcpyDeviceToHost);

    printf("Printing final results...\n");
	printfinal();
	printf("\nDone.\n\n");
	
	return 0;
}